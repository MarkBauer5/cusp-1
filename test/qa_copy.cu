/* -*- c++ -*- */
/*
 * Copyright 2021 Josh Morman
 *
 * SPDX-License-Identifier: LGPL-3.0-or-later
 *
 */

#include <complex>
#include <cusp/copy.cuh>
#include <cusp/cusp.cuh>
#include <gtest/gtest.h>

template <typename T> void run_test(int N)
{
    std::vector<std::complex<float>> host_input_data(N);
    for (int i = 0; i < N; i++) {
      host_input_data[i] = std::complex<float>(i, -i);
    }
    std::vector<std::complex<float>> host_output_data(N);
  
    void *dev_input_data;
    void *dev_output_data;
  
    hipMalloc(&dev_input_data, N * sizeof(std::complex<float>));
    hipMalloc(&dev_output_data, N * sizeof(std::complex<float>));
  
    hipMemcpy(dev_input_data, host_input_data.data(),
               N * sizeof(std::complex<float>), hipMemcpyHostToDevice);
  
    int ncopies = N * sizeof(std::complex<float>) / sizeof(T);
    cusp::copy<T> op;
    
    op.launch_default_occupancy({dev_input_data}, {dev_output_data}, ncopies);
  
    hipDeviceSynchronize();
    hipMemcpy(host_output_data.data(), dev_output_data,
               N * sizeof(std::complex<float>), hipMemcpyDeviceToHost);
  
    EXPECT_EQ(host_input_data, host_output_data);
}

TEST(CopyKernel, Basic) {
  int N = 1024 * 1000;

  run_test<uint64_t>(N);
  run_test<uint8_t>(N);
}
